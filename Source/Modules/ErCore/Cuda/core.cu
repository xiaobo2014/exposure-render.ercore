/*
*	@file
*	@author  Thomas Kroes <t.kroes at tudelft.nl>
*	@version 1.0
*	
*	@section LICENSE
*	
*	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
*	
*	Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
*	Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
*	Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
*
*	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <map>

using namespace std;

texture<unsigned short, 3, hipReadModeNormalizedFloat> TexVolume0;
texture<unsigned short, 3, hipReadModeNormalizedFloat> TexVolume1;

#include "color.h"

map<int, int> gTracersHashMap;
map<int, int> gVolumesHashMap;
map<int, int> gObjectsHashMap;
map<int, int> gClippingObjectsHashMap;
map<int, int> gTexturesHashMap;
map<int, int> gBitmapsHashMap;

//CONSTANT_DEVICE float gDensityScale			= 0.0f;
//CONSTANT_DEVICE float gStepFactorPrimary	= 0.0f;
//CONSTANT_DEVICE float gStepFactorShadow		= 0.0f;

CONSTANT_DEVICE float gDensityScale = 10.0f;
CONSTANT_DEVICE float gStepFactorPrimary = 0.0078125f;
CONSTANT_DEVICE float gStepFactorShadow = 0.01953125f;

//DEVICE float gDensityScale = 10.0f;
//DEVICE float gStepFactorPrimary = 0.0078125f;
//DEVICE float gStepFactorShadow = 0.01953125f;

#include "statistics.h"
#include "tracer.h"
#include "volume.h"
#include "object.h"
#include "texture.h"
#include "bitmap.h"

//#include "wrapper.cuh"

int VolumeNum = 1;


DEVICE ExposureRender::Tracer*			gpTracer			= NULL;
//ExposureRender::Cuda::Allocate(gpTracer, VolumeNum);
DEVICE ExposureRender::Volume* 			gpVolumes			= NULL;
DEVICE ExposureRender::Object*			gpObjects			= NULL;
DEVICE ExposureRender::Texture*			gpTextures			= NULL;
DEVICE ExposureRender::Bitmap*			gpBitmaps			= NULL;


/*
DEVICE ExposureRender::Tracer			global_Tracer;
DEVICE ExposureRender::Volume			global_Volumes;
DEVICE ExposureRender::Object			global_Objects;
DEVICE ExposureRender::Texture			global_Textures;
DEVICE ExposureRender::Bitmap			global_Bitmaps;

DEVICE ExposureRender::Tracer*			gpTracer = &global_Tracer;
DEVICE ExposureRender::Volume* 		gpVolumes = &global_Volumes;
DEVICE ExposureRender::Object*			gpObjects = &global_Objects;
DEVICE ExposureRender::Texture*		gpTextures = &global_Textures;
DEVICE ExposureRender::Bitmap*			gpBitmaps = &global_Bitmaps;
*/

#include "list.cuh"

ExposureRender::Cuda::List<ExposureRender::Tracer, ExposureRender::HostTracer>					gTracers;
ExposureRender::Cuda::List<ExposureRender::Volume, ExposureRender::HostVolume>					gVolumes;
ExposureRender::Cuda::List<ExposureRender::Object, ExposureRender::HostObject>					gObjects;
ExposureRender::Cuda::List<ExposureRender::Texture, ExposureRender::HostTexture>				gTextures;
ExposureRender::Cuda::List<ExposureRender::Bitmap, ExposureRender::HostBitmap>					gBitmaps;


#include "autofocus.cuh"
#include "render.cuh"

namespace ExposureRender
{

EXPOSURE_RENDER_DLL void BindTracer(const HostTracer& Tracer, const bool& Bind /*= true*/)
{
	if (Bind)
		gTracers.Bind(Tracer);
	else
		gTracers.Unbind(Tracer);

	gTracersHashMap = gTracers.HashMap;
}

EXPOSURE_RENDER_DLL void BindVolume(const HostVolume& Volume, const bool& Bind /*= true*/)
{
	if (Bind)
		gVolumes.Bind(Volume);
	else
		gVolumes.Unbind(Volume);

	gVolumesHashMap = gVolumes.HashMap;
}

EXPOSURE_RENDER_DLL void BindObject(const HostObject& Object, const bool& Bind /*= true*/)
{
	if (Bind)
		gObjects.Bind(Object);
	else
		gObjects.Unbind(Object);

	gObjectsHashMap = gObjects.HashMap;
}

EXPOSURE_RENDER_DLL void BindTexture(const HostTexture& Texture, const bool& Bind /*= true*/)
{
	if (Bind)
		gTextures.Bind(Texture);
	else
		gTextures.Unbind(Texture);

	gTexturesHashMap = gTextures.HashMap;
}

EXPOSURE_RENDER_DLL void BindBitmap(const HostBitmap& Bitmap, const bool& Bind /*= true*/)
{
	if (Bind)
		gBitmaps.Bind(Bitmap);
	else
		gBitmaps.Unbind(Bitmap);

	gBitmapsHashMap = gBitmaps.HashMap;
}

EXPOSURE_RENDER_DLL void Render(int TracerID, Statistics& Statistics)
{
	hipEvent_t EventStart, EventStop;

	Cuda::HandleCudaError(hipEventCreate(&EventStart));
	Cuda::HandleCudaError(hipEventCreate(&EventStop));
	Cuda::HandleCudaError(hipEventRecord(EventStart, 0));

	Tracer& Tracer = gTracers[TracerID];

    //old code
	//const float DensityScale		= Tracer.VolumeProperty.GetDensityScale();
	//const float StepFactorPrimary	= gVolumes[gVolumesHashMap[Tracer.VolumeIDs[0]]].MinStep * Tracer.VolumeProperty.GetStepFactorPrimary();
	//const float StepFactorShadow	= gVolumes[gVolumesHashMap[Tracer.VolumeIDs[0]]].MinStep * Tracer.VolumeProperty.GetStepFactorShadow();
	
	//Cuda::HostToConstantDevice(&DensityScale, "gDensityScale");
	//Cuda::HostToConstantDevice(&StepFactorPrimary, "gStepFactorPrimary");
	//Cuda::HostToConstantDevice(&StepFactorShadow, "gStepFactorShadow");

    //new code
    float DensityScale = Tracer.VolumeProperty.GetDensityScale();
    float StepFactorPrimary = gVolumes[gVolumesHashMap[Tracer.VolumeIDs[0]]].MinStep * Tracer.VolumeProperty.GetStepFactorPrimary();
    float StepFactorShadow = gVolumes[gVolumesHashMap[Tracer.VolumeIDs[0]]].MinStep * Tracer.VolumeProperty.GetStepFactorShadow();

    //Cuda::MemCopyHostToDevice(&DensityScale, &gDensityScale);
    //Cuda::MemCopyHostToDevice(&StepFactorPrimary, &gStepFactorPrimary);
    //Cuda::MemCopyHostToDevice(&StepFactorShadow, &gStepFactorShadow);
	
	/*
	if (Tracer.NoEstimates == 0)
	{
		if (Tracer.Camera.FocusMode == Enums::AutoFocus)
		{
			float AutoFocusDistance = -1.0f;

			const Vec2i FilmUV((int)(Tracer.Camera.FocusUV[0] * (float)Tracer.FrameBuffer.Resolution[0]), (int)(Tracer.Camera.FocusUV[1] * (float)Tracer.FrameBuffer.Resolution[1]));
			ComputeAutoFocusDistance(FilmUV, AutoFocusDistance, Statistics);

			if (AutoFocusDistance >= 0.0f)
				Tracer.Camera.FocalDistance = AutoFocusDistance;
		}
	}
	*/

	gTracers.Synchronize(TracerID);

	if (Tracer.VolumeIDs[0] >= 0)
		gVolumes[Tracer.VolumeIDs[0]].Voxels.Bind(TexVolume0);

	if (Tracer.VolumeIDs[1] >= 0)
		gVolumes[Tracer.VolumeIDs[1]].Voxels.Bind(TexVolume1);

    Volume& Volume = gVolumes[Tracer.VolumeIDs[0]];

    Object& Object = gObjects[0];

    Texture& Texture = gTextures[0];

	Render(Tracer, Volume, Object, Texture, Statistics);
		
	if (Tracer.NoiseReduction)
		BilateralFilterRunningEstimate(Tracer, Statistics);
	
	Tracer.NoEstimates++;

	Cuda::HandleCudaError(hipEventRecord(EventStop, 0));
	Cuda::HandleCudaError(hipEventSynchronize(EventStop));
																							
	float TimeDelta = 0.0f;
																							
	Cuda::HandleCudaError(hipEventElapsedTime(&TimeDelta, EventStart, EventStop), "hipEventElapsedTime");
	
	Statistics.SetStatistic("FPS", "%.1f", "frames/sec", 1000.0f / TimeDelta);
														
	Cuda::HandleCudaError(hipEventDestroy(EventStart));
	Cuda::HandleCudaError(hipEventDestroy(EventStop));										

}

EXPOSURE_RENDER_DLL void GetDisplayEstimate(int TracerID, ColorRGBAuc* pData)
{
	FrameBuffer& FB = gTracers[TracerID].FrameBuffer;

    //origin code
	Cuda::MemCopyDeviceToHost(FB.DisplayEstimate.GetData(), (ColorRGBAuc*)pData, FB.DisplayEstimate.GetNoElements());

    //lxb update
    //memcpy((void*)pData, (void*)FB.DisplayEstimate.GetData(), FB.Resolution[0] * FB.Resolution[1] * sizeof(ColorRGBAuc));
    
}

}
